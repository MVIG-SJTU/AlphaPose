#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

#define THREADS_PER_BLOCK 1024

inline int GET_BLOCKS(const int N) {
  int optimal_block_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int max_block_num = 65000;
  return min(optimal_block_num, max_block_num);
}

template <typename scalar_t>
__device__ scalar_t bilinear_interpolate(const scalar_t *bottom_data,
                                         const int height, const int width,
                                         scalar_t y, scalar_t x) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    return 0;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (scalar_t)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (scalar_t)x_low;
  } else {
    x_high = x_low + 1;
  }

  scalar_t ly = y - y_low;
  scalar_t lx = x - x_low;
  scalar_t hy = 1. - ly;
  scalar_t hx = 1. - lx;
  // do bilinear interpolation
  scalar_t lt = bottom_data[y_low * width + x_low];
  scalar_t rt = bottom_data[y_low * width + x_high];
  scalar_t lb = bottom_data[y_high * width + x_low];
  scalar_t rb = bottom_data[y_high * width + x_high];
  scalar_t w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  scalar_t val = (w1 * lt + w2 * rt + w3 * lb + w4 * rb);

  return val;
}

template <typename scalar_t>
__global__ void ROIAlignForward(const int nthreads, const scalar_t *bottom_data,
                                const scalar_t *bottom_rois,
                                const scalar_t spatial_scale,
                                const int sample_num, const int channels,
                                const int height, const int width,
                                const int pooled_height, const int pooled_width,
                                scalar_t *top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the aligned output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const scalar_t *offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    scalar_t roi_start_w = offset_bottom_rois[1] * spatial_scale;
    scalar_t roi_start_h = offset_bottom_rois[2] * spatial_scale;
    scalar_t roi_end_w = (offset_bottom_rois[3] + 1) * spatial_scale;
    scalar_t roi_end_h = (offset_bottom_rois[4] + 1) * spatial_scale;

    // Force malformed ROIs to be 1x1
    scalar_t roi_width = fmaxf((scalar_t)roi_end_w - roi_start_w, 0.);
    scalar_t roi_height = fmaxf((scalar_t)roi_end_h - roi_start_h, 0.);

    scalar_t bin_size_h = roi_height / pooled_height;
    scalar_t bin_size_w = roi_width / pooled_width;

    const scalar_t *offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    int sample_num_h = (sample_num > 0)
                           ? sample_num
                           : ceil(roi_height / pooled_height);  // e.g., = 2
    int sample_num_w =
        (sample_num > 0) ? sample_num : ceil(roi_width / pooled_width);

    scalar_t h = (scalar_t)(ph + 0.5) * bin_size_h + roi_start_h;
    scalar_t w = (scalar_t)(pw + 0.5) * bin_size_w + roi_start_w;

    int hstart = fminf(floor(h), height - 2);
    int wstart = fminf(floor(w), width - 2);

    scalar_t output_val = 0;
    for (int iy = 0; iy < sample_num_h; iy++) {
      const scalar_t y = roi_start_h + ph * bin_size_h +
                         (scalar_t)(iy + scalar_t(.5f)) * bin_size_h /
                             (scalar_t)(sample_num_h);
      for (int ix = 0; ix < sample_num_w; ix++) {
        const scalar_t x = roi_start_w + pw * bin_size_w +
                           (scalar_t)(ix + scalar_t(.5f)) * bin_size_w /
                               (scalar_t)(sample_num_w);
        scalar_t val = bilinear_interpolate<scalar_t>(offset_bottom_data,
                                                      height, width, y, x);
        output_val += val;
      }
    }
    output_val /= (sample_num_h * sample_num_w);
    top_data[index] = output_val;
  }
}

int ROIAlignForwardLaucher(const at::Tensor features, const at::Tensor rois,
                           const float spatial_scale, const int sample_num,
                           const int channels, const int height,
                           const int width, const int num_rois,
                           const int pooled_height, const int pooled_width,
                           at::Tensor output) {
  const int output_size = num_rois * pooled_height * pooled_width * channels;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.scalar_type(), "ROIAlignLaucherForward", ([&] {
        const scalar_t *bottom_data = features.data_ptr<scalar_t>();
        const scalar_t *rois_data = rois.data_ptr<scalar_t>();
        scalar_t *top_data = output.data_ptr<scalar_t>();

        ROIAlignForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data, rois_data, scalar_t(spatial_scale),
                sample_num, channels, height, width, pooled_height,
                pooled_width, top_data);
      }));
  THCudaCheck(hipGetLastError());
  return 1;
}

template <typename scalar_t>
__device__ void bilinear_interpolate_gradient(const int height, const int width,
                                              scalar_t y, scalar_t x,
                                              scalar_t &w1, scalar_t &w2,
                                              scalar_t &w3, scalar_t &w4,
                                              int &x_low, int &x_high,
                                              int &y_low, int &y_high) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (scalar_t)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (scalar_t)x_low;
  } else {
    x_high = x_low + 1;
  }

  scalar_t ly = y - y_low;
  scalar_t lx = x - x_low;
  scalar_t hy = 1. - ly;
  scalar_t hx = 1. - lx;

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename scalar_t>
__global__ void ROIAlignBackward(
    const int nthreads, const scalar_t *top_diff, const scalar_t *bottom_rois,
    const scalar_t spatial_scale, const int sample_num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, scalar_t *bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the aligned output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const scalar_t *offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    scalar_t roi_start_w = offset_bottom_rois[1] * spatial_scale;
    scalar_t roi_start_h = offset_bottom_rois[2] * spatial_scale;
    scalar_t roi_end_w = (offset_bottom_rois[3] + 1) * spatial_scale;
    scalar_t roi_end_h = (offset_bottom_rois[4] + 1) * spatial_scale;

    // Force malformed ROIs to be 1x1
    scalar_t roi_width = fmaxf((scalar_t)roi_end_w - roi_start_w, 0.);
    scalar_t roi_height = fmaxf((scalar_t)roi_end_h - roi_start_h, 0.);

    scalar_t bin_size_h = roi_height / pooled_height;
    scalar_t bin_size_w = roi_width / pooled_width;

    scalar_t *offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;
    int offset_top = (n * channels + c) * pooled_height * pooled_width +
                     ph * pooled_width + pw;
    scalar_t offset_top_diff = top_diff[offset_top];

    int sample_num_h = (sample_num > 0)
                           ? sample_num
                           : ceil(roi_height / pooled_height);  // e.g., = 2
    int sample_num_w =
        (sample_num > 0) ? sample_num : ceil(roi_width / pooled_width);

    const scalar_t count = (scalar_t)(sample_num_h * sample_num_w);

    scalar_t h = (scalar_t)(ph + 0.5) * bin_size_h + roi_start_h;
    scalar_t w = (scalar_t)(pw + 0.5) * bin_size_w + roi_start_w;

    int hstart = fminf(floor(h), height - 2);
    int wstart = fminf(floor(w), width - 2);

    for (int iy = 0; iy < sample_num_h; iy++) {
      const scalar_t y =
          roi_start_h + ph * bin_size_h +
          (scalar_t)(iy + .5f) * bin_size_h / (scalar_t)(sample_num_h);
      for (int ix = 0; ix < sample_num_w; ix++) {
        const scalar_t x =
            roi_start_w + pw * bin_size_w +
            (scalar_t)(ix + .5f) * bin_size_w / (scalar_t)(sample_num_w);
        scalar_t w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient<scalar_t>(
            height, width, y, x, w1, w2, w3, w4, x_low, x_high, y_low, y_high);
        scalar_t g1 = offset_top_diff * w1 / count;
        scalar_t g2 = offset_top_diff * w2 / count;
        scalar_t g3 = offset_top_diff * w3 / count;
        scalar_t g4 = offset_top_diff * w4 / count;
        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(offset_bottom_diff + y_low * width + x_low, g1);
          atomicAdd(offset_bottom_diff + y_low * width + x_high, g2);
          atomicAdd(offset_bottom_diff + y_high * width + x_low, g3);
          atomicAdd(offset_bottom_diff + y_high * width + x_high, g4);
        }
      }
    }
  }
}

int ROIAlignBackwardLaucher(const at::Tensor top_grad, const at::Tensor rois,
                            const float spatial_scale, const int sample_num,
                            const int channels, const int height,
                            const int width, const int num_rois,
                            const int pooled_height, const int pooled_width,
                            at::Tensor bottom_grad) {
  const int output_size = num_rois * pooled_height * pooled_width * channels;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "ROIAlignLaucherBackward", ([&] {
        const scalar_t *top_diff = top_grad.data_ptr<scalar_t>();
        const scalar_t *rois_data = rois.data_ptr<scalar_t>();
        scalar_t *bottom_diff = bottom_grad.data_ptr<scalar_t>();
        if (sizeof(scalar_t) == sizeof(double)) {
          fprintf(stderr, "double is not supported\n");
          exit(-1);
        }

        ROIAlignBackward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, top_diff, rois_data, spatial_scale, sample_num,
                channels, height, width, pooled_height, pooled_width,
                bottom_diff);
      }));
  THCudaCheck(hipGetLastError());
  return 1;
}
